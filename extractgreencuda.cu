#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA kernel
__global__
void vecCompare(int *R, int *G, int *B, int *result, int n)     //A is for the green array
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        if(G[i] > 2 * R[i] || G[i] > 2 * B[i])
        {
            result[i] = 1;
        }
        else
        {
            result[i] = 0;
        }
    }
}

extern "C"
void compareMatrices(int height, int width, int*r, int*g, int*b, int*green)
{
    // Size of vectors
    int n = height * width;

    // Device input vectors
    int *d_r;
    int *d_g;
    int *d_b;
    //Device output vector
    int *d_green;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(int);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_r, bytes);
    hipMalloc(&d_g, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_green, bytes);

    // Copy host vectors to device
    hipMemcpy( d_r, r, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_g, g, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecCompare<<<gridSize, blockSize>>>(d_r, d_g, d_b, d_green, n);

    // Copy array back to host
    hipMemcpy( green, d_green, bytes, hipMemcpyDeviceToHost );

    // Release device memory
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
    hipFree(d_green);

    // Release host memory
    free(r);
    free(g);
    free(b);
}
